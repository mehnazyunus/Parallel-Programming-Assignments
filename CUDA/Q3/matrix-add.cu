#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define N 10
#define TILE_SIZE 16.0


__global__ void add_matrix(int* dev_mat1, int* dev_mat2, int* dev_sum_mat, int nrow, int ncol)
{
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	
	if(row<nrow && col<ncol)
	dev_sum_mat[row*ncol+col]=dev_mat1[row*ncol+col]+dev_mat2[row*ncol+col];
	
}

void printMatrix(int* mat,int ncol)
{
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		printf("%d ",mat[i*ncol+j]);
	    
		printf("\n");
	}
}

int main()
{	
	int nrow = 100,ncol = 100;
	
	int *mat1,*mat2,*sum_mat;

	int size=nrow*ncol*sizeof(int);
	mat1 = (int *)malloc(size);
	mat2 = (int *)malloc(size);
	sum_mat = (int *)malloc(size);
	
	srand(time(NULL));
	
	for(int i=0;i<nrow;i++)
		for(int j=0;j<ncol;j++)
		{
			mat1[i*ncol+j] = rand()%4+1;
			mat2[i*ncol+j] = rand()%5+1;
		}

	
	int* dev_mat1, *dev_mat2, *dev_sum_mat;
	
	hipMalloc((void**)&dev_mat1,size);
	hipMalloc((void**)&dev_mat2,size);
	hipMalloc((void**)&dev_sum_mat,size);
	
	hipMemcpy(dev_mat1,mat1,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2,mat2,size,hipMemcpyHostToDevice);
	
	dim3 dimBlock(TILE_SIZE,TILE_SIZE);
	dim3 dimGrid(ceil(ncol/TILE_SIZE),ceil(nrow/TILE_SIZE));
	add_matrix<<<dimGrid,dimBlock>>>(dev_mat1,dev_mat2,dev_sum_mat,nrow,ncol);
	
	hipMemcpy(sum_mat,dev_sum_mat,size,hipMemcpyDeviceToHost);
	printf("Matrix 1\n");
	printMatrix(mat1,ncol);
	printf("Matrix 2\n");
	printMatrix(mat2,ncol);
	printf("Matrix Sum for first N x N matrix\n");
	printMatrix(sum_mat,ncol);
	
	hipFree(dev_mat1);
	hipFree(dev_mat2);
	hipFree(dev_sum_mat);

	free(mat1);
	free(mat2);
	free(sum_mat);	
}
