#include "hip/hip_runtime.h"
/*
  Joe Antony - 15CO220
  Prateek Kembhavi - 15CO223 
*/

#include "wb.h"
#include <stdio.h>
#define TILE_WIDTH 16

__global__ void matrixMultiply(float * A, float * B, float * C,
  		       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;
    float Pvalue = 0;

    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
          ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
       else
          ds_M[ty][tx] = 0;
       if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
          ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
       else
          ds_N[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += ds_M[ty][k] * ds_N[k][tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; 
    float * hostB; 
    float * hostC; 
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows;
    int numAColumns;
    int numBRows; 
    int numBColumns;
    int numCRows; 
    int numCColumns;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    
    numCRows = numARows;
    numCColumns = numBColumns;
    
    hostC = (float *)malloc(sizeof(float) * numCRows * numCColumns);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    
    hipMalloc(&deviceA, sizeof(float) * numARows * numAColumns);
    hipMalloc(&deviceB, sizeof(float) * numBRows * numBColumns);
    hipMalloc(&deviceC, sizeof(float) * numCRows * numCColumns);

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    
    hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice);

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    
    dim3 dimGrid((numCColumns-1)/TILE_WIDTH+1, (numCRows-1)/TILE_WIDTH+1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    
    matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC,
                                          numARows, numAColumns,
                                          numBRows, numBColumns,
                                          numCRows, numCColumns);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    
    hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
