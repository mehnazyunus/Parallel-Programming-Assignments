#include "hip/hip_runtime.h"
#include "wb.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
#define NUM_BINS 4096


__global__ void hist_kernel(unsigned int *deviceInput, unsigned int *deviceBins,unsigned int inputLength)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  
 if(i<inputLength)
    {int item = deviceInput[i];  atomicAdd(&(deviceBins[item]),1);}
 
}

__global__ void bins_cap(unsigned int *deviceBins)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  deviceBins[i] = min(127,deviceBins[i]);
}

int main(int argc, char *argv[]) {

 unsigned  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

 
  wbArg_t args = wbArg_read(argc,argv);
  FILE* inp = fopen(argv[1],"r");
	fscanf(inp,"%d",&inputLength);

  wbTime_start(Generic, "Importing data and creating memory on host");
	
  hostInput = new unsigned int[inputLength];
  
  for(int i=0;i<inputLength;i++)
  {
    fscanf(inp,"%d",&hostInput[i]);
  }
  
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(int));
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  wbTime_start(GPU, "Allocating GPU memory.");
  
  cout<<inputLength<<endl;
  int size = inputLength*sizeof(int);

  hipMalloc((void **)&deviceInput,size);
  hipMalloc((void **)&deviceBins,NUM_BINS*sizeof(unsigned int));
  hipMemset(deviceBins,0,NUM_BINS*sizeof(unsigned int));
  CUDA_CHECK(hipDeviceSynchronize());

  wbTime_stop(GPU, "Allocating GPU memory.");
  wbTime_start(GPU, "Copying input memory to the GPU.");

  hipMemcpy(deviceInput,hostInput,size,hipMemcpyHostToDevice);
  
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  wbLog(TRACE, "Launching kernel");
  wbTime_start(Compute, "Performing CUDA computation");
  
  int threads = 1024;
  hist_kernel<<<(inputLength-1)/1024+1, threads>>>(deviceInput,deviceBins,inputLength);
  bins_cap<<<4,1024>>>(deviceBins);
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  
  hipMemcpy(hostBins,deviceBins,NUM_BINS*sizeof(unsigned int),hipMemcpyDeviceToHost);
  CUDA_CHECK(hipDeviceSynchronize());

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");

  hipFree(deviceInput);
  hipFree(deviceBins);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostBins, NUM_BINS);
  free(hostBins);
  free(hostInput);
  return 0;
}
