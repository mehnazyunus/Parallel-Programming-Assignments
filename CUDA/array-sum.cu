#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void get_sum(float *dev_arr,float *dev_sum)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	atomicAdd(dev_sum,dev_arr[idx]);
}
int main()
{
	float *dev_arr, *dev_sum , *arr,sum = 0;
	int N;
	
	printf("Enter Array size : \n");
	scanf(" %d",&N);

	arr = (float *)malloc(N*sizeof(float));
	//Filling the array
	for(int i=0;i<N;++i)
		arr[i] = 1.0;

	hipMalloc((void **)&dev_arr,N*sizeof(float));
	hipMalloc((void **)&dev_sum,sizeof(float));

	hipMemcpy(dev_arr,arr,N*sizeof(float),hipMemcpyHostToDevice);
	
	int size = ceil(sqrt(N));
	get_sum<<<size,size>>>(dev_arr,dev_sum);

	hipMemcpy(&sum,&dev_sum,sizeof(float),hipMemcpyDeviceToHost);
	printf("The sum of the array is %f\n",sum );

	hipFree(dev_arr);
	hipFree(dev_sum);
	hipFree(dev_N);
	return 0;
}