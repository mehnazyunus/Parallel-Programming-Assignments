#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
	hipDeviceProp_t prop;

	int count;
	hipGetDeviceCount(&count);

	for(int i=0;i<count;++i)
	{
		hipGetDeviceProperties(&prop,i);
		printf( "--- General Information for device %d ---\n", i );
		printf( "Name:%s\n", prop.name );
		printf( "Compute capability:%d.%d\n", prop.major, prop.minor );
		printf( "Clock rate:%d\n", prop.clockRate );
		printf( "Device copy overlap:" );
		
		if (prop.deviceOverlap)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );
		
		printf( "Kernel execution timeout :" );
		
		if (prop.kernelExecTimeoutEnabled)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );
		
		printf( "--- Memory Information for device %d ---\n", i );
		printf( "Total global mem:%ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem:%ld\n", prop.totalConstMem );
		
		printf( "Max mem pitch:%ld\n", prop.memPitch );
		printf( "Texture Alignment:%ld\n", prop.textureAlignment );
	}
}
