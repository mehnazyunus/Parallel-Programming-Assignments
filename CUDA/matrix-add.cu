#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define TILE_SIZE 16

void printMatrix(int *mat,int nrow,int ncol)
{
	for(int i=0;i<nrow;i++)
	{
		for(int j=0;j<ncol;j++)
			printf("%d ",mat[i*ncol+j]);
		printf("\n");
	}
}
__global__ void add_matrix(int *dev_mat1,int *dev_mat2,int *dev_sum_mat,int nrow,int ncol)
{
	//printf("Hello\n");
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	int idx = x + y*ncol;
	printf("Dimensions %d %d\n",nrow,ncol );
	if(x < ncol && y < nrow )
		dev_sum_mat[idx] = dev_mat1[idx] + dev_mat2[idx];
}

int main()
{
	int nrow=10,ncol=10;
	int size = nrow*ncol*sizeof(int);
	int *mat1,*mat2,*sum_mat;

	mat1 = (int *)malloc(size);
	mat2 = (int *)malloc(size);
	sum_mat = (int *)malloc(size);

	srand(NULL);

	for(int i=0;i<nrow;i++)
		for(int j=0;j<ncol;j++)
		{
			mat1[i*ncol+j] = rand()%4+1;
			mat2[i*ncol+j] = rand()%5+1;
		}

	int *dev_mat1,*dev_mat2,*dev_sum_mat;

	hipMalloc((void **)&dev_mat1,size);
	hipMalloc((void **)&dev_mat2,size);
	hipMalloc((void **)&dev_sum_mat,size);


	hipMemcpy(dev_mat1,mat1,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_mat2,mat2,size,hipMemcpyHostToDevice);


	dim3 dimBlock(TILE_SIZE,TILE_SIZE);
	dim3 dimGrid(ceil(ncol/TILE_SIZE),ceil(nrow/TILE_SIZE));

	add_matrix<<<dimGrid,dimBlock>>>(dev_mat1,dev_mat2,dev_sum_mat,nrow,ncol);

	hipMemcpy(sum_mat,dev_sum_mat,size,hipMemcpyDeviceToHost);



	printf("Matrix 1\n");
	printMatrix(mat1,nrow,ncol);
	printf("Matrix 2\n");
	printMatrix(mat2,nrow,ncol);
	printf("Matrix Sum\n");
	printMatrix(sum_mat,nrow,ncol);

	hipFree(dev_mat1);
	hipFree(dev_mat2);
	hipFree(dev_sum_mat);

	free(mat1);
	free(mat2);
	free(sum_mat);
}